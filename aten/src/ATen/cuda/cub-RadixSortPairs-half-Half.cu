#include <ATen/cuda/cub-RadixSortPairs.cuh>

namespace at::cuda::hipcub::detail {

AT_INSTANTIATE_SORT_PAIRS_8(decltype(::c10::impl::ScalarTypeToCPPType<at::ScalarType::Half>::t), at::ScalarType::Half)

} // namespace at::cuda::hipcub::detail
